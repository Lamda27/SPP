#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out)
{
	//int thread = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread = x + gridDim.x * blockDim.x * y;

	if(thread >= width*height)
		return;

	int pixel_out = thread;
	int pixel_in = 3*thread;
	image_out[pixel_out] = image[pixel_in] * 0.0722f + image[pixel_in + 1] * 0.7152f + image[pixel_in + 2] * 0.2126f;

}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
//TODO: Define the cGaussian array on the constant memory (2 pt)

void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	//TODO: Copy computed fGaussian to the cGaussian on device memory (2 pts)
	float cGaussian[64];
	hipMemset(cGaussian, 0, 64);
	hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, 64 * sizeof(float), 0, hipMemcpyHostToDevice);
}

//TODO: implement cuda_gaussian() kernel (3 pts)
__device__
inline double cuda_gaussian(float x, double sigma){
	return expf(-(powf(x, 2)) / (2 * powf(sigma, 2)));
}

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
	// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size r
	float fGaussian[64];
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread = x + gridDim.x * blockDim.x * y;

	if(thread >= width*height)
		return;

	double iFiltered = 0;
	double wP = 0;
	unsigned char centrePx = input[thread];

	for (int dy = -r; dy <= r; dy++) {
		int neighborY = y+dy;
		if (neighborY < 0)
			neighborY = 0;
		else if (neighborY >= height)
			neighborY = height - 1;
		for (int dx = -r; dx <= r; dx++) {
			int neighborX = x+dx;
			if (neighborX < 0)
				neighborX = 0;
			else if (neighborX >= width)
				neighborX = width - 1;
			int currPx_idx = neighborX + gridDim.x * blockDim.x * neighborY;
			unsigned char currPx = input[currPx_idx];
			double w = (fGaussian[dy+r] * fGaussian[dx+r]) * cuda_gaussian(centrePx - currPx, sI);
			iFiltered += x * currPx;
			wP += x;
		}
	}
output[thread] = iFiltered / wP;
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	printf("Starting gpu_pipeline.\n");
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
	BYTE *d_input = NULL;
	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************
	printf("Launching grayscale kernel.\n");
	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale);

        int block_dim_x, block_dim_y;
        block_dim_x = (int) sqrt(suggested_blockSize);
				block_dim_y = (int) sqrt(suggested_blockSize);

        dim3 gray_block(block_dim_x, block_dim_y); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2 pts
				int grid_dim_x, grid_dim_y;
				grid_dim_x = (int) sqrt(suggested_minGridSize);
				grid_dim_y = (int) sqrt(suggested_minGridSize);
				dim3 gray_grid(grid_dim_x, grid_dim_y);

        // Allocate the intermediate image buffers for each step
        Image img_out(input.cols, input.rows, 1, "P5");
				printf("Test.\n");
        for (int i = 0; i < 2; i++)
        {
            //TODO: allocate memory on the device (2 pts)
            //TODO: intialize allocated memory on device to zero (2 pts)
							hipMalloc(&d_image_out[i], image_size * sizeof(BYTE));
							hipMemset(d_image_out[i], 0, image_size * sizeof(BYTE));
        }

        //copy input image to device
        //TODO: Allocate memory on device for input image (2 pts)
				hipMalloc(&d_input, image_size * sizeof(BYTE));
				hipMemset(d_input, 0, image_size * sizeof(BYTE));
        //TODO: Copy input image into the device memory (2 pts)
				hipMemcpy(d_input, &input, image_size * sizeof(BYTE), hipMemcpyHostToDevice);

        hipEventRecord(start, 0); // start timer
        // Convert input image to grayscale
        //TODO: Launch cuda_grayscale() (2 pts)
				hipDeviceSynchronize();
				cuda_grayscale<<<gray_grid, gray_block>>>(input.cols, input.rows, d_input, d_image_out[0]);

        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
				hipEventElapsedTime(&time, start, stop);
				cout << "GPU Grayscaling time: " << time << " (ms)\n";
				cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;

        //TODO: transfer image from device to the main memory for saving onto the disk (2 pts)
				hipMemcpy(d_image_out[0], &img_out, image_size * sizeof(BYTE), hipMemcpyDeviceToHost);

        savePPM(img_out, "image_gpu_gray.ppm");


	// ******* Bilateral filter kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter);

        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize);

        dim3 bilateral_block(block_dim_x,block_dim_y); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2pts
				grid_dim_x = grid_dim_y = (int) sqrt(suggested_minGridSize);
				dim3 bilateral_grid(grid_dim_x, grid_dim_y);

        // Create gaussain 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
	//TODO: Launch cuda_bilateral_filter() (2 pts)
				cuda_bilateral_filter<<<bilateral_grid, bilateral_block>>>(d_image_out[0], d_image_out[1],
					input.cols, input.cols, r, sI, sS);

        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host
	//TODO: transfer image from device to the main memory for saving onto the disk (2 pts)
				hipMemcpy(d_image_out[1], &img_out, image_size * sizeof(BYTE), hipMemcpyDeviceToHost);
				savePPM(img_out, "image_gpu_bilateral.ppm");

        // ************** Finalization, cleaning up ************
				printf("Device Error:\t%s\n", hipGetErrorString(hipGetLastError()));

        // Free GPU variables
	//TODO: Free device allocated memory (3 pts)
				hipFree(d_input);
				hipFree(d_image_out[0]);
				hipFree(d_image_out[1]);
}
